#include "hip/hip_runtime.h"
#include "aesp.h"

__global__ void setup_kernel(hiprandState *globalstate, unsigned long seed)
{
	    int id = threadIdx.x + blockIdx.x*blockDim.x;
	        if (id<MAXPSIZE)
			    hiprand_init(seed, id, 0, &globalstate[id]);
}


/**************************************************************
* For each cell determine if new particles should be created
**************************************************************/
__global__ void evolve_c_state(particle *d_elements, cell *d_cell, hiprandState *globalState, int *d_states, int *d_index, int *d_active)
{
    int id =  threadIdx.x+blockDim.x*blockIdx.x;
    if (id<CELLX*CELLY){
	    cell *c = &d_cell[id];
	    if (atomicCAS(&c->busy,0,1)> 0){
		    if (RADIATE == 0 && c->radiant == 0){
			    hiprandState state = globalState[id];
			    double d = hiprand_uniform_double(&state);
			    if (atomicCAS(d_index,0,*d_index)>0){
				    if (d < c->cre){
					    int j = atomicSub(d_index,1);
					    int r = atomicAdd(d_active,1);
					    int idc = d_states[j+1];
					    d_states[j+1] = -1;
					    particle *p = &d_elements[idc];
					    p->alive = 1;
					    d = hiprand_uniform_double(&state);
					    p->x = d*DIMX/CELLX+c->x0*DIMX/CELLX;
					    d = hiprand_uniform_double(&state);
					    p->y = d*DIMY/CELLY+c->y0*DIMY/CELLY;
//					    p->speed = 0;
					    atomicAdd(&c->pcount,1);
					    //				    int recind = p->x/(int)(DIMX/CELLX)+ (p->y/(int)(DIMY/CELLY)*CELLY);
					    //    				    printf("EC Particle %i from Block %i (has now %i particles) with x=%i, y=%i is reactivated, reconstructed cell id is %i dead index is %i\n",idc, id, c->pcount,p->x, p->y, recind, *d_index );
				    }
				    atomicCAS(&c->busy,1,0);
			    }
		    } else {
			    for (int m = 0; m < c->emissivity; m++){
				    hiprandState state = globalState[id];
				    double d = hiprand_uniform_double(&state);
				    if (atomicCAS(d_index,0,*d_index)>0){
					    int j = atomicSub(d_index,1);
					    int r = atomicAdd(d_active,1);
					    int idc = d_states[j+1];
					    d_states[j+1] = -1;
					    particle *p = &d_elements[idc];
					    p->alive = 1;
					    d = hiprand_uniform_double(&state);
					    p->x = d*DIMX/CELLX+c->x0*DIMX/CELLX;
					    d = hiprand_uniform_double(&state);
					    p->y = d*DIMY/CELLY+c->y0*DIMY/CELLY;
//					    p->speed = 0;
					    atomicAdd(&c->pcount,1);
				    }
			    }
    		    }
	    }
    }
}

/**************************************************************
* For each particle determine if it should die (be absorbed)
**************************************************************/
__global__ void evolve_p_state(particle *d_elements, cell *d_cell, hiprandState *globalState, int *d_states, int *d_index, int *d_active)
{
    	int id =  threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<MAXPSIZE){
		particle *p = &d_elements[id];
		if (atomicCAS(&p->alive,1,p->alive)> 0){
			int xcell = p->x/(int)(DIMX/CELLX);
			int ycell = p->y/(int)(DIMY/CELLY);
			int cellid = xcell+ycell*CELLY;

//			printf("EP Particle %i at %i and %i from Block %i with x=%i, y=%i d index is %i\n",p->id, p->x, p->y, cellid, d_cell[cellid].x0, d_cell[cellid].y0, *d_index);
			hiprandState state = globalState[id];
			double d = hiprand_uniform_double(&state);
			if (d > (d_cell[cellid]).ann){
				int pc = atomicSub(&(d_cell[cellid]).pcount,1); 
    				atomicSub(&p->alive,1);
    				int r = atomicSub(d_active,1);
    				int j = atomicAdd(d_index,1);
    				d_states[j] = p->id;
				p->x = 0;
				p->y = 0;
//				p->speed = 0;
//				printf("EP Particle %i from Block %i with x=%i, y=%i died,  d index is %i, counter is now %i\n",p->id, cellid, d_cell[cellid].x0, d_cell[cellid].y0, *d_index, pc);
			}
		}
	}

}

/**************************************************************
* Moving the particle in some direction, blah blah blah
**************************************************************/
__global__ void propagate(Lock lock, particle *d_elements, cell *d_cell, hiprandState *globalState, int *d_states, int *d_index, int *d_active)
{
    	int id =  threadIdx.x+blockDim.x*blockIdx.x;
    	if (id<MAXPSIZE){
    		particle *p = &d_elements[id];
		int xcell0 = p->x/(int)(DIMX/CELLX);
		int ycell0 = p->y/(int)(DIMY/CELLY);
		int cellid0 = xcell0+ycell0*CELLY;

    		int xcell1;
    		int ycell1;
		int cellid1;
//		double tempdir = 0;
    		if (atomicCAS(&p->alive,1,p->alive)> 0){
			printf("Index %i of PRP Particle %i at %i and %i with angle %f from Block %i with x=%i, y=%i and id=%i d index is %i\n",id,p->id, p->x, p->y, p->dir, cellid0, d_cell[cellid0].x0, d_cell[cellid0].y0, d_cell[cellid0].id, *d_index);
//    			if (p->speed == 0){
//  			} else {
    				p->x = (int)(cos(p->dir)*p->s)+p->x;
    				p->y = (int)(sin(p->dir)*p->s)+p->y;
				xcell1 = p->x/(int)(DIMX/CELLX);
				ycell1 = p->y/(int)(DIMY/CELLY);
//    				if (p->x > p->xf || p->y > p->yf)
//    					p->speed = 0;
//    			}
			printf("PRP Particle %i at %i and %i with angle %f\n",p->id, p->x, p->y, p->dir);
			cellid1 = xcell1+ycell1*CELLY;
    			if (p->x >DIMX || p->x < 0 || p->y > DIMY || p->y < 0){
				if (PERMEABILITY < 1){
					if (p->x >DIMX){
						if (p->dir > ((double)(4/3)/(double)(PI))){
							p->dir = (4/3*PI)-(p->dir-((4/3)*PI));	
							p->x = DIMX-(p->x-DIMX);
						} else if (p->dir <((double)PI/(double)2)){
							p->dir = PI-p->dir;	
							p->x = DIMX-(p->x-DIMX);
						}
					} else if (p->x < 0){
						if (p->dir > PI){
							p->dir = 2*PI-(p->dir-PI);
							p->x = -p->x;
						} else if (p->dir > ((double)PI/(double)(2))){
							p->dir = PI-p->dir;
							p->x = -p->x;
						}
					}
					if (p->y > DIMY){
						if (p->dir < ((double)PI/(double)(2))){
							p->dir = (2*PI)-p->dir;
							p->y = DIMY-(p->y-DIMY);
						} else if (p->dir < PI){
							p->dir = PI+(PI-p->dir);
							p->y = DIMY-(p->y-DIMY);
						}
					} else if (p->y < 0){
						if (p->dir < ((double)(4/3)/(double)(PI))){
							p->dir = PI+(p->dir-PI);
							p->y = -p->y;
						} else if (p->dir < ((double)2*PI)){
							p->dir = (PI/2)+(p->dir-((4/3)*PI));
							p->y = -p->y;
						}
					}
//					printf("Reflected Particle %i at %i and %i with angle %f from Block %i with x=%i, y=%i and id=%i d index is %i\n",p->id, p->x, p->y, p->dir,cellid0, d_cell[cellid0].x0, d_cell[cellid0].y0, d_cell[cellid0].id, *d_index);
				} else {
					atomicSub(&p->alive,1);
					int r = atomicSub(d_active,1);
					int j = atomicAdd(d_index,1);
					atomicSub(&d_cell[cellid0].pcount,1);
					d_states[j] = p->id;
					printf("PRP Particle %i at %i and %i from Block %i with x=%i, y=%i dnd id=%i  index is %i died by exit from field\n",p->id, p->x, p->y, cellid1, d_cell[cellid1].x0, d_cell[cellid1].y0, d_cell[cellid1].id, *d_index);
				}
			}else if (xcell0 != xcell1 || ycell0!=ycell1){
				atomicSub(&d_cell[cellid0].pcount,1);
				atomicAdd(&d_cell[cellid1].pcount,1);
				if (INTERACTION > 0){
	    				hiprandState state = globalState[id];
    					double d = hiprand_uniform_double(&state);
    					p->dir = (d*2*PI);
				}
//    				p->s =p->speed*ST;
  //  				p->xf = (int)(cos(p->dir)*(double)S)+p->x;
    //				p->yf = (int)(sin(p->dir)*(double)S)+p->y;
    //				p->x = (int)(cos(p->dir)*p->s)+p->x;
    //				p->y = (int)(sin(p->dir)*p->s)+p->y;
//				xcell1 = p->x/(int)(DIMX/CELLX);
//				ycell1 = p->y/(int)(DIMY/CELLY);

				printf("PRP Particle %i crossed from Block %i with x=%i, y=%i to block %i with x=%i, y=%i\n",p->id, cellid0, d_cell[cellid0].x0, d_cell[cellid0].y0, cellid1, d_cell[cellid1].x0, d_cell[cellid1].y0);
			}
    		}
    	}
}



int init_particle(particle *p, int  x, int y, int id){

	p->x = x;
	p->y = y;
	p->id = id;
	p->dir = ((double)rand()/(double)RAND_MAX)*360;
	p->speed = 0;
	p->s = V*ST;
	p->alive = 1;
	return 0;
}



int init_cell(cell *c, int a , int b, int *h_states, particle *p){

	c->x0 = a;
	c->y0 = b;
	c->id = a+b*CELLY;
	c->pcount = 0;
	c->emissivity = EMS;
	c->cre = ((double)rand()/(double)RAND_MAX)/c->emissivity;
	c->ann = ((double)rand()/(double)RAND_MAX)/c->emissivity;
	c->radiant = 0;
	if (radiate_x == a && radiate_y == b && RADIATE > 0){
		c->radiant = 1;
		for (int v = 0; v < c->emissivity;v++){
			int x = a*DIMX/CELLX+rand()%(int)(DIMX/CELLX);
			int y = b*DIMY/CELLY+rand()%(int)(DIMY/CELLY);
			if (h_index >=1){
			h_states[h_index] = -1;
			h_index--;
			c->pcount++;
			init_particle(&p[h_index],x,y,h_index);
//		int x0 = p[h_index].x/(int)(DIMX/CELLX);
//		int y0 = p[h_index].y/(int)(DIMY/CELLY);
//		int cellid = x0+y0*CELLY;
//		printf("Particle %i from Block %i (is a=%i and b=%i and has now %i particles) was created at x=%i, y=%i, extracted cell id=%i\n",h_index, c->id, a,b, c->pcount,x, y, cellid);
			}
		}
	} else if (RADIATE < 1){
		for (int v = 0; v < c->emissivity;v++){
			int x = a*DIMX/CELLX+rand()%(int)(DIMX/CELLX);
			int y = b*DIMY/CELLY+rand()%(int)(DIMY/CELLY);
			if (h_index >=1){
			h_states[h_index] = -1;
			h_index--;
			c->pcount++;
			init_particle(&p[h_index],x,y,h_index);
//		int x0 = p[h_index].x/(int)(DIMX/CELLX);
//		int y0 = p[h_index].y/(int)(DIMY/CELLY);
//		int cellid = x0+y0*CELLY;
//		printf("Particle %i from Block %i (is a=%i and b=%i and has now %i particles) was created at x=%i, y=%i, extracted cell id=%i\n",h_index, c->id, a,b, c->pcount,x, y, cellid);
			}
		}
	}
	return 0;
}

int destroy_particle(particle *p){
	free(p);
	return 0;

}

int main(){



	h_index = MAXPSIZE;
	int a,b = 0;
	int memSize = MAXPSIZE*sizeof(hiprandState);
	hipMalloc((void**) &globalstate, memSize);

	//initialize an array of pointers to particles
	memSize = MAXPSIZE*sizeof(particle);
	hipMalloc((void**) &d_elements, memSize);

	//initialize an array of inactive particles indices
	memSize = MAXPSIZE*sizeof(int);
	hipMalloc((void**) &d_states, memSize);

	//initialize an semaphore for counting inactive particles
	memSize = sizeof(int);
	hipMalloc(&d_index, memSize);

	//initialize an semaphore for counting inactive particles
	memSize = sizeof(int);
	hipMalloc(&d_total, memSize);
	hipMalloc(&d_active, memSize);
	hipMalloc(&d_residual, memSize);


	//initialize an array of spatial cells
	memSize = CELLX*CELLY*sizeof(cell);
	hipMalloc((void**) &d_cells, memSize);

	//initialize h_states
	h_states = new int[MAXPSIZE];
	for (int b = 0; b < MAXPSIZE;b++)
		h_states[b] = MAXPSIZE-b-1;

		printf("States Done\n");
	//initialize all cells - space locations
	srand((unsigned)time(NULL));
	h_cells = new cell[(int)(CELLX*CELLY)];
	h_elements = new particle[MAXPSIZE];
	srand((unsigned)time(NULL));
//	double angle = 0;
//	int xx = 0;
//	int yy = 0;
	for (int d = 0; d < (int)(CELLX*CELLY);d++){
		init_cell(&h_cells[d],a,b,h_states,h_elements);
		a++;
		if (a >= CELLX){a=0; b++;}
//		printf("Angle %f x0=%i and y0=%i",angle, 0,0);
//	xx = (int)(cos(angle)*(double)S);
  //  	yy = (int)(sin(angle)*(double)S);
	
//		printf(" and xx=%i yy=%i\n",xx,yy);
//		angle += (double)(2*PI)/(double)(CELLX*CELLY);
//		printf("Cell %i (is a=%i and b=%i and has now %i particles and %i index) \n",d, a,b, h_cells[d].pcount,h_cells[d].id);
	}
		//printf("Cells Done %i, %i \n", h_index, MAXPSIZE);
//	for (int a = MAXPSIZE-1; a >= MAXPSIZE-10;a--){
//		printf("Particle %i at coordinates %i, %i is alive %d\n", h_elements[a].id,h_elements[a].x, h_elements[a].y, h_elements[a].alive);
//	}

	h_total = MAXPSIZE;
	h_active = MAXPSIZE-h_index;
	h_residual = 0.0;


	//allocate space for cells and particles on cuda
	memSize = sizeof(int);
	printf("Initial particle count is: %i\n",h_active);
	printf("Initial particle count is: %i\n",h_index);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_total, &h_total, memSize, hipMemcpyHostToDevice)));
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_active, &h_active, memSize, hipMemcpyHostToDevice)));
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_residual, &h_residual, memSize, hipMemcpyHostToDevice)));
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_index, &h_index, memSize, hipMemcpyHostToDevice)));
	memSize = MAXPSIZE*sizeof(int);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_states, h_states, memSize, hipMemcpyHostToDevice)));
	memSize = CELLX*CELLY*sizeof(cell);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_cells, h_cells, memSize, hipMemcpyHostToDevice)));
	memSize = MAXPSIZE*sizeof(particle);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(d_elements, h_elements, memSize, hipMemcpyHostToDevice)));
	//hipDeviceSynchronize();

	for (int p = 0; p< 1000; p++){
		propagate<<<PBLOCKS, PBLOCKS>>>(lock, d_elements, d_cells, globalstate, d_states, d_index, d_active);
		setup_kernel <<<PBLOCKS, PBLOCKS >>>(globalstate, time(NULL));
		if  (ANNHILATION > 0){
			evolve_p_state<<<PBLOCKS, PBLOCKS>>>(d_elements, d_cells, globalstate, d_states, d_index, d_active);
		}
		setup_kernel <<<PBLOCKS, PBLOCKS >>>(globalstate, time(NULL));
	//	evolve_c_state<<<CBLOCKS, CBLOCKS>>>(d_elements, d_cells, globalstate, d_states, d_index, d_active);
	}

	memSize = MAXPSIZE*sizeof(particle);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(h_elements, d_elements, memSize, hipMemcpyDeviceToHost)));
	memSize = sizeof(int);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(&h_active, d_active, memSize, hipMemcpyDeviceToHost)));
	memSize = MAXPSIZE*sizeof(int);
	printf("CUDA error: %s\n", hipGetErrorString(hipMemcpy(h_states, d_states, memSize, hipMemcpyDeviceToHost)));

	printf("Final particle count is: %i\n",h_active);

	for (int a = MAXPSIZE-1; a >= MAXPSIZE-10;a--){
		printf("Particle %i at coordinates %i, %i is alive %d\n", h_elements[a].id,h_elements[a].x, h_elements[a].y, h_elements[a].alive);
	}

return 0;
}
